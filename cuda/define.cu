/**
 * File Name : define.cu
 * Author : Yang Fan
 * Date : 2018/11/27
 * define some functions
 */

#pragma once

#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>

/**
 * define a cuda call
 */
#define CUDA_CALL(x) { \
	const hipError_t e = (x);\
	if(e != hipSuccess)		\
	{		\
		printf("\nCUDA Error : %s (err_num = %d)\n", hipGetErrorString(e), e); \
		hipDeviceReset(); \
		assert(0); \
	}\
}

/**
 * define screen width & height & isFullScreen
 */
#define SCREEN_WIDTH 1024
#define SCREEN_HEIGHT 768
#define IS_FULL_SCREEN false

/*
 * print device information 
 */
#define PRINT_DEVICE_INFORMATION() { \
    int dc;   \
	hipGetDeviceCount(&dc); \
	if (dc == 0) { \
		printf("error : no device supporting cuda\n"); \
		exit(1); \
	} \
		\
	int dev = 0;\
	hipSetDevice(dev);	\
	hipDeviceProp_t devProps;	\
	hipGetDeviceProperties(&devProps, dev); \
	\
	printf("name : %s \ntotalGlobalMem : %zdM\n" , devProps.name , devProps.totalGlobalMem >> 20); \
}