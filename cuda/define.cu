/**
 * File Name : define.cuh
 * Author : Yang Fan
 * Date : 2018/11/27
 * define some functions with cuda
 */

#pragma once
#include <hip/hip_runtime.h>


/**
 * define a cuda call
 */
#define CUDA_CALL(x) { \
	const hipError_t e = (x);	\
	if (e != hipSuccess) \							\
	{							\
		printf("\nCUDA Error : %s (err_num = %d)\n" , hipGetErrorString(e) , e);					\
		hipDeviceReset();						\
		assert(0);							\
	}                  \
}
