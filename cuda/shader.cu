#include "hip/hip_runtime.h"
/**
 * File Name : shader.cu
 * Author : Yang Fan
 * Date : 2019/1/6
 * defined shader
 */

#pragma once
#include <>
#include "../common/define.h"
#include "../common/texture.h"
#include <vector>


__device__ Color& TexSampler2D(const Texture& texture, unsigned char* texturesPixels , const Math::Vector2& uv) {

	Color color;

	const int tx = uv._x * texture.width;
	const int ty = uv._y * texture.height;

	auto idx = (ty * texture.width + tx) * 4;

	CLAMP(idx, 4, texture.width * texture.height * 4);

	color.a = texturesPixels[idx - 1] / 255.0f;
	color.b = texturesPixels[idx - 2] / 255.0f;
	color.g = texturesPixels[idx - 3] / 255.0f;
	color.r = texturesPixels[idx - 4] / 255.0f;

	return color;
}

/*
 * Pixel Shader
 */
__device__ Color& PixelShader(Pixel& pixel , const Texture& texture , unsigned char* texturesPixels) {
	// ������
	const auto ambient = 0.1;

	auto texColor = TexSampler2D(texture, texturesPixels, pixel.uv);

	const auto directionLight = Math::Vector3(0 , 1 , -1).normalize();
	const auto normal = pixel.normal.normalize();

	auto nd = Math::Vector3::dot(directionLight , normal);

	CLAMP01(nd);

	auto color = texColor * (ambient + nd);

	return color;
}

__global__ void KernelPixelShader(Color* colors, Pixel* pixels, Texture* textures, unsigned char* texturesPixels,
                                  const int numElements) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numElements) {
		colors[idx] = PixelShader(pixels[idx] , textures[0] , texturesPixels);
	}
}

extern "C" void CallPixelShader(const std::vector<Pixel>& pixels, const std::vector<Texture>& textures,
                                std::vector<Color>& colors) {
	const int numPixels = pixels.size();
	const int numTextures = textures.size();

	Pixel* dPixels;
	CUDA_CALL(hipMalloc(&dPixels , sizeof(Pixel) * numPixels));
	CUDA_CALL(hipMemset(dPixels , 0 , sizeof(Pixel) * numPixels));
	Color* dColors;
	CUDA_CALL(hipMalloc(&dColors , sizeof(Color) * numPixels));
	CUDA_CALL(hipMemset(dColors , 0 , sizeof(Color) * numPixels));
	Texture* dTextures;
	CUDA_CALL(hipMalloc(&dTextures , sizeof(Texture) * numTextures));
	CUDA_CALL(hipMemset(dTextures , 0 , sizeof(Texture) * numTextures));

	// ���¿�����һ������� pixels ����
	unsigned char* dTexturesPixels;
	CUDA_CALL(hipMalloc(&dTexturesPixels, sizeof(unsigned char) * textures[0].width * textures[0].height * 4));
	CUDA_CALL(hipMemcpy(dTexturesPixels, textures[0].pixels, sizeof(unsigned char) * textures[0].width * textures[0].height * 4, hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(dPixels , &pixels[0] , numPixels * sizeof(Pixel) , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dTextures , &textures[0] , numTextures * sizeof(Texture) , hipMemcpyHostToDevice))

	// 64
	KernelPixelShader<<<(numPixels + 15) / 16 , 16>>>(dColors, dPixels, dTextures, dTexturesPixels, numPixels);

	CUDA_CALL(hipMemcpy(&colors[0] , dColors , numPixels * sizeof(Color) , hipMemcpyDeviceToHost));

	CUDA_CALL(hipFree(dPixels));
	CUDA_CALL(hipFree(dColors));
	CUDA_CALL(hipFree(dTextures));
	CUDA_CALL(hipFree(dTexturesPixels));
}

 