#include "hip/hip_runtime.h"
/**
 * File Name : device.cu
 * Author : Yang Fan
 * Date : 2019/4/2
 * define device mixed
 */

#pragma once
#include <>
#include "../common/device.h"
#include "../common/define.h"
#include "../common/texture.h"
#include <vector>

/*
 * ������ɫ
 */
__device__ void SetPixel(int x, int y, const Color& color, Uint8* pixelColors, int screenWidth, int screenHeight) {
	auto r = color.r;
	auto g = color.g;
	auto b = color.b;
	auto a = color.a;
	CLAMP01(a);
	CLAMP01(b);
	CLAMP01(g);
	CLAMP01(r);

	auto i = (y * screenWidth + x) * 4;

	CLAMP(i , 0 , screenWidth * screenHeight * 4 - 1);

	pixelColors[i - 1] = static_cast<Uint8>(a * 255);
	pixelColors[i - 2] = static_cast<Uint8>(b * 255);
	pixelColors[i - 3] = static_cast<Uint8>(g * 255);
	pixelColors[i - 4] = static_cast<Uint8>(r * 255);
}

/*
 * ��Ȳ���
 */
__device__ void TestDepth(int x , int y , float depth , float* depths , bool& isSuccess , int screenWidth , int screenHeight) {
	const auto idx = y * screenWidth + x;
	const auto cdp = depths[idx];

	if (cdp == 0 || depth <= cdp) {
		depths[idx] = depth;
		isSuccess = true;
	}
}


/*
 * ��Ⱦ���߻�Ͻ׶�
 */
__global__ void KernelMixed(Pixel* pixels , Color* colors , Uint8* pixelColors , float* depths , int screenWidth , int screenHeight , int numElements) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < numElements) {

		const int x = pixels[idx].pos._x;
		const int y = pixels[idx].pos._y;

		auto isFirst = false;
		TestDepth(x , y , pixels[idx].pos._z , depths , isFirst , screenWidth , screenHeight);
		if (isFirst) {
			SetPixel(x , y , colors[idx] , pixelColors , screenWidth , screenHeight);
		}
	}
}


extern "C" void CallMixed(std::vector<Pixel>& pixels, std::vector<Color>& colors , Uint8* pixelColors , float *depths , int screenWidth , int screenHeight) {
	const int numPixels = pixels.size();
	const int screenPixelSize = screenWidth * screenHeight;
	
	Pixel* dPixels;
	CUDA_CALL(hipMalloc(&dPixels , sizeof(Pixel) * numPixels));
	CUDA_CALL(hipMemset(dPixels , 0 , sizeof(Pixel) * numPixels));
	Color* dColors;
	CUDA_CALL(hipMalloc(&dColors , sizeof(Color) * numPixels));
	CUDA_CALL(hipMemset(dColors , 0 , sizeof(Color) * numPixels));
	Uint8* dPixelColors;
	CUDA_CALL(hipMalloc(&dPixelColors , sizeof(Uint8) * screenPixelSize * 4));
	CUDA_CALL(hipMemset(dPixelColors , 0 , sizeof(Uint8) * screenPixelSize * 4));
	float* dDepths;
	CUDA_CALL(hipMalloc(&dDepths , sizeof(float) * screenPixelSize));
	CUDA_CALL(hipMemset(dDepths , 0 , sizeof(float) * screenPixelSize));

	CUDA_CALL(hipMemcpy(dPixelColors , pixelColors , sizeof(Uint8) * screenPixelSize  * 4 , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dDepths , depths , sizeof(float) * screenPixelSize , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dPixels , &pixels[0] , sizeof(Pixel) * numPixels , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dColors , &colors[0] , sizeof(Color) * numPixels , hipMemcpyHostToDevice));

	// ��ˮ��ִ��

	KernelMixed<<<(numPixels + 15) / 16 , 16>>>(dPixels , dColors , dPixelColors , dDepths , screenWidth , screenHeight , numPixels);

	CUDA_CALL(hipMemcpy(depths , dDepths , sizeof(float) * screenPixelSize , hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(pixelColors , dPixelColors , sizeof(Uint8) * screenPixelSize  * 4, hipMemcpyDeviceToHost));
	
	CUDA_CALL(hipFree(dDepths));
	CUDA_CALL(hipFree(dPixels));
	CUDA_CALL(hipFree(dColors));
	CUDA_CALL(hipFree(dPixelColors));
}

