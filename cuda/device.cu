#include "hip/hip_runtime.h"
/**
 * File Name : device.cu
 * Author : Yang Fan
 * Date : 2019/4/2
 * define device mixed
 */

#pragma once
#include <>
#include "../common/device.h"
#include "../common/define.h"
#include "../common/texture.h"
#include <vector>

__device__ void intersect(const Ray& ray, const Triangle& triangle, IntersectResult& intersectResult) {
	const auto origin = ray.origin;
	const auto direction = ray.direction;

	const auto u = triangle.top.pos3D - triangle.btm.pos3D;
	const auto v = triangle.mid.pos3D - triangle.btm.pos3D;
	const auto norm = Math::Vector3::cross(u, v).normalize();

	const auto b = Math::Vector3::dot(norm, direction);

	if (fabs(b) < 0) return;

	const auto w0 = origin - triangle.btm.pos3D;

	const auto a = - Math::Vector3::dot(norm, w0);

	const auto r = a / b;
	if (r <= 0.0f) return;

	intersectResult.intersectPoint = origin + direction * r;
	intersectResult.distance = r;

	const auto uu = Math::Vector3::dot(u, u);
	const auto uv = Math::Vector3::dot(u, v);
	const auto vv = Math::Vector3::dot(v, v);

	const auto w = intersectResult.intersectPoint - triangle.btm.pos3D;

	const auto wu = Math::Vector3::dot(w, u);
	const auto wv = Math::Vector3::dot(w, v);

	const auto d = uv * uv - uu * vv;

	const auto s = (uv * wv - vv * wu) / d;
	if (s < 0.0f || s > 1.0f) return;

	const auto t = (uv * wu - uu * wv) / d;
	if (t < 0.0f || (s + t) > 1.0f) return;


	intersectResult.isSucceed = true;
}

/*
 * ������ɫ
 */
__device__ void SetPixel(int x, int y, const Color& color, Uint8* pixelColors, int screenWidth, int screenHeight) {
	auto r = color.r;
	auto g = color.g;
	auto b = color.b;
	auto a = color.a;
	CLAMP01(a);
	CLAMP01(b);
	CLAMP01(g);
	CLAMP01(r);

	auto i = (y * screenWidth + x) * 4;
	const auto size = (screenWidth - 1) * (screenHeight - 1) * 4;

	CLAMP(i , 4 , size);

	pixelColors[i - 1] = static_cast<Uint8>(a * 255);
	pixelColors[i - 2] = static_cast<Uint8>(b * 255);
	pixelColors[i - 3] = static_cast<Uint8>(g * 255);
	pixelColors[i - 4] = static_cast<Uint8>(r * 255);
}

/*
 * ������ɫ
 */
__device__ void GetPixel(int x, int y, Uint8* pixelColors, int screenWidth, int screenHeight, Color& color) {
	auto i = (y * screenWidth + x) * 4;
	const auto size = (screenWidth - 1) * (screenHeight - 1) * 4;

	CLAMP(i , 4 , size);

	color.a = pixelColors[i - 1] / static_cast<float>(255);
	color.b = pixelColors[i - 2] / static_cast<float>(255);
	color.g = pixelColors[i - 3] / static_cast<float>(255);
	color.r = pixelColors[i - 4] / static_cast<float>(255);
}

/*
 * ��Ȳ���
 */
__device__ void TestDepth(int x, int y, float depth, float* depths, bool& isSuccess, int screenWidth,
                          int screenHeight) {
	if (x >= screenWidth || x <= 0 || y <= 0 || y >= screenHeight) return;

	const auto idx = y * screenWidth + x;
	const auto cdp = depths[idx];

	if (cdp == 0 || depth <= cdp) {
		depths[idx] = depth;
		isSuccess = true;
	}
}

/*
 * �������
 */
__device__ void SampleLight(Pixel& pixel, Triangle* triangles, Color& color, int numTriangles) {
	if (pixel.sType == LIGHT) return;

	const auto lightOrigin = Math::Vector3(0, 2, 0);

	const auto lightDirection = (lightOrigin - pixel.pos3D).normalize();

	const auto distance = (lightOrigin - pixel.pos3D).length();

	const Math::Vector3 normal = pixel.normal.normalize();

	// ������Ӱ
	Ray ray;
	ray.isActive = true;
	ray.origin = pixel.pos3D;
	ray.direction = lightDirection;

	bool isShadow = false;

#pragma unroll
	for (auto i = 0; i < numTriangles; ++i) {
		IntersectResult iTmp{false};
		intersect(ray, triangles[i], iTmp);
		if (iTmp.isSucceed && iTmp.distance > 0.05f && iTmp.distance < distance - 0.05) {
			isShadow = true;
			break;
		}
	}

	// �������	
	float ambient = 0.3;

	float nd = Math::Vector3::dot(lightDirection, normal);
	CLAMP01(nd);

	color = color * (ambient + nd * !isShadow);
}

/*
 * ���㷴��
 */
__device__ void SampleReflect(Pixel& pixel, Triangle* triangles, Color& color, int numTriangles) {
	// ������Ӱ
	Ray ray;
	ray.isActive = true;
	ray.origin = pixel.pos3D;
	ray.direction = pixel.normal.normalize();

	float minDistance = INT_MAX;
	//
	// IntersectResult itRet;
	// int idxTriangle = 0;
#pragma unroll
	for (auto i = 0; i < numTriangles; ++i) {
		IntersectResult iTmp{false};
		intersect(ray, triangles[i], iTmp);
		if (iTmp.isSucceed && iTmp.distance < minDistance) {
			minDistance = iTmp.distance;
			// itRet = iTmp;
			// idxTriangle = i;
			if (triangles[i].mid.sType == LIGHT) color = Color::white();
			else color = triangles[i].mid.color;
		}
	}

	// const auto pos3D = itRet.intersectPoint;

	// float x3DMin, x3DMax, y3DMin, y3DMax;
	// float x2DMin, x2DMax, y2DMin, y2DMax;
	//
	// /*X�������С*/
	// if (triangles[idxTriangle].btm.pos3D._x > triangles[idxTriangle].top.pos3D._x) {
	// 	if (triangles[idxTriangle].btm.pos3D._x > triangles[idxTriangle].mid.pos3D._x) {
	// 		x3DMax = triangles[idxTriangle].btm.pos3D._x;
	// 		x2DMax = triangles[idxTriangle].btm.pos._x;
	// 	}
	// 	else {
	// 		x3DMax = triangles[idxTriangle].mid.pos3D._x;
	// 		x2DMax = triangles[idxTriangle].mid.pos._x;
	// 	}
	// }
	// else {
	// 	if (triangles[idxTriangle].top.pos3D._x > triangles[idxTriangle].mid.pos3D._x) {
	// 		x3DMax = triangles[idxTriangle].top.pos3D._x;
	// 		x2DMax = triangles[idxTriangle].top.pos._x;
	// 	}
	// 	else {
	// 		x3DMax = triangles[idxTriangle].mid.pos3D._x;
	// 		x2DMax = triangles[idxTriangle].mid.pos._x;
	// 	}
	// }
	//
	//
	// if (triangles[idxTriangle].btm.pos3D._x < triangles[idxTriangle].top.pos3D._x) {
	// 	if (triangles[idxTriangle].btm.pos3D._x < triangles[idxTriangle].mid.pos3D._x) {
	// 		x3DMin = triangles[idxTriangle].btm.pos3D._x;
	// 		x2DMin = triangles[idxTriangle].btm.pos._x;
	// 	}
	// 	else {
	// 		x3DMin = triangles[idxTriangle].mid.pos3D._x;
	// 		x2DMin = triangles[idxTriangle].mid.pos._x;
	// 	}
	// }
	// else {
	// 	if (triangles[idxTriangle].top.pos3D._x < triangles[idxTriangle].mid.pos3D._x) {
	// 		x3DMin = triangles[idxTriangle].top.pos3D._x;
	// 		x2DMin = triangles[idxTriangle].top.pos._x;
	// 	}
	// 	else {
	// 		x3DMin = triangles[idxTriangle].mid.pos3D._x;
	// 		x2DMin = triangles[idxTriangle].mid.pos._x;
	// 	}
	// }
	//
	// /*Y�������С*/
	// if (triangles[idxTriangle].btm.pos3D._y > triangles[idxTriangle].top.pos3D._y) {
	// 	if (triangles[idxTriangle].btm.pos3D._y > triangles[idxTriangle].mid.pos3D._y) {
	// 		y3DMax = triangles[idxTriangle].btm.pos3D._y;
	// 		y2DMax = triangles[idxTriangle].btm.pos._y;
	// 	}
	// 	else {
	// 		y3DMax = triangles[idxTriangle].mid.pos3D._y;
	// 		y2DMax = triangles[idxTriangle].mid.pos._y;
	// 	}
	// }
	// else {
	// 	if (triangles[idxTriangle].top.pos3D._y > triangles[idxTriangle].mid.pos3D._y) {
	// 		y3DMax = triangles[idxTriangle].top.pos3D._y;
	// 		y2DMax = triangles[idxTriangle].top.pos._y;
	// 	}
	// 	else {
	// 		y3DMax = triangles[idxTriangle].mid.pos3D._y;
	// 		y2DMax = triangles[idxTriangle].mid.pos._y;
	// 	}
	// }
	//
	//
	// if (triangles[idxTriangle].btm.pos3D._y < triangles[idxTriangle].top.pos3D._y) {
	// 	if (triangles[idxTriangle].btm.pos3D._y < triangles[idxTriangle].mid.pos3D._y) {
	// 		y3DMin = triangles[idxTriangle].btm.pos3D._y;
	// 		y2DMin = triangles[idxTriangle].btm.pos._y;
	// 	}
	// 	else {
	// 		y3DMin = triangles[idxTriangle].mid.pos3D._y;
	// 		y2DMin = triangles[idxTriangle].mid.pos._y;
	// 	}
	// }
	// else {
	// 	if (triangles[idxTriangle].top.pos3D._y < triangles[idxTriangle].mid.pos3D._y) {
	// 		y3DMin = triangles[idxTriangle].top.pos3D._y;
	// 		y2DMin = triangles[idxTriangle].top.pos._y;
	// 	}
	// 	else {
	// 		y3DMin = triangles[idxTriangle].mid.pos3D._y;
	// 		y2DMin = triangles[idxTriangle].mid.pos._y;
	// 	}
	// }
	//
	// auto dx = 0.0f, dy = 0.0f;
	// if (x3DMax - x3DMin != 0.0f)
	// 	dx = (pos3D._x - x3DMin) / (x3DMax - x3DMin);
	// if (y3DMax - y3DMin != 0.0f)
	// 	dy = (pos3D._y - y3DMin) / (y3DMax - y3DMin);
	//
	// const auto x = static_cast<int>(x2DMin + (x2DMax - x2DMin) * dx);
	// const auto y = static_cast<int>(y2DMin + (y2DMax - y2DMin) * dy);
	//
	// GetPixel(x, y, pixelColors, screenWidth, screenHeight, color);

}


/*
 * ��Ⱦ���߻�Ͻ׶�
 */
__global__ void KernelMixedReflect(Pixel* pixels, Color* colors, Triangle* triangles, Uint8* pixelColors, float* depths,
                                   int screenWidth, int screenHeight, int numTriangles, int numElements) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numElements) {

		const int x = pixels[idx].pos._x;
		const int y = pixels[idx].pos._y;

		auto isFirst = false;
		TestDepth(x, y, pixels[idx].pos._z, depths, isFirst, screenWidth, screenHeight);
		if (isFirst) {

			/*�������*/
			auto lightColor = Color::white();
			SampleLight(pixels[idx], triangles, lightColor, numTriangles);

			/*���㷴��*/
			auto reflectColor = Color::black();
			SampleReflect(pixels[idx], triangles, reflectColor, numTriangles);

			/*�����ɫ*/
			colors[idx] = (colors[idx] * (1 - pixels[idx].reflectiveness) + reflectColor * pixels[idx].reflectiveness) *
				lightColor;

			/*��ɫ*/
			SetPixel(x, y, colors[idx], pixelColors, screenWidth, screenHeight);
		}
	}
}


/*
 * ��Ⱦ���߻�Ͻ׶�
 */
__global__ void KernelMixed(Pixel* pixels, Color* colors, Uint8* pixelColors, float* depths,
                            int screenWidth, int screenHeight, int numElements) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numElements) {

		const int x = pixels[idx].pos._x;
		const int y = pixels[idx].pos._y;

		auto isFirst = false;
		TestDepth(x, y, pixels[idx].pos._z, depths, isFirst, screenWidth, screenHeight);
		if (isFirst) {
			/*��ɫ*/
			SetPixel(x, y, colors[idx], pixelColors, screenWidth, screenHeight);
		}
	}
}


extern "C" void CallMixed(std::vector<Pixel>& pixels, std::vector<Color>& colors,
                          const std::vector<Triangle>& triangles, Uint8* pixelColors, float* depths, int screenWidth,
                          int screenHeight) {
	if (pixels.empty()) return;

	const int numPixels = pixels.size();
	const int numTriangles = triangles.size();
	const int screenPixelSize = screenWidth * screenHeight;

	Pixel* dPixels;
	CUDA_CALL(hipMalloc(&dPixels , sizeof(Pixel) * numPixels));
	CUDA_CALL(hipMemset(dPixels , 0 , sizeof(Pixel) * numPixels));
	Color* dColors;
	CUDA_CALL(hipMalloc(&dColors , sizeof(Color) * numPixels));
	CUDA_CALL(hipMemset(dColors , 0 , sizeof(Color) * numPixels));
	Uint8* dPixelColors;
	CUDA_CALL(hipMalloc(&dPixelColors , sizeof(Uint8) * screenPixelSize * 4));
	CUDA_CALL(hipMemset(dPixelColors , 0 , sizeof(Uint8) * screenPixelSize * 4));
	float* dDepths;
	CUDA_CALL(hipMalloc(&dDepths , sizeof(float) * screenPixelSize));
	CUDA_CALL(hipMemset(dDepths , 0 , sizeof(float) * screenPixelSize));

	Triangle* dTriangles;
	CUDA_CALL(hipMalloc(&dTriangles , sizeof(Triangle) * numTriangles));
	CUDA_CALL(hipMemset(dTriangles , 0 , sizeof(Triangle) * numTriangles));


	CUDA_CALL(hipMemcpy(dPixelColors , pixelColors , sizeof(Uint8) * screenPixelSize * 4 , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dDepths , depths , sizeof(float) * screenPixelSize , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dPixels , &pixels[0] , sizeof(Pixel) * numPixels , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dColors , &colors[0] , sizeof(Color) * numPixels , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dTriangles , &triangles[0] , sizeof(Triangle) * numTriangles , hipMemcpyHostToDevice));
	//
	// // ��ˮ��ִ��
	//
	// // ��һ����ɫ
	// KernelMixed<<<(numPixels + 255) / 256 , 256>>>(dPixels, dColors, dPixelColors, dDepths, screenWidth,
	//                                                screenHeight, numPixels);
	//
	// hipDeviceSynchronize();

	// �ڶ�����ɫ
	KernelMixedReflect<<<(numPixels + 255) / 256 , 256>>>(dPixels, dColors, dTriangles, dPixelColors, dDepths,
	                                                      screenWidth,
	                                                      screenHeight, numTriangles, numPixels);

	hipDeviceSynchronize();

	CUDA_CALL(hipMemcpy(depths , dDepths , sizeof(float) * screenPixelSize , hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(pixelColors , dPixelColors , sizeof(Uint8) * screenPixelSize * 4, hipMemcpyDeviceToHost));

	CUDA_CALL(hipFree(dDepths));
	CUDA_CALL(hipFree(dPixels));
	CUDA_CALL(hipFree(dColors));
	CUDA_CALL(hipFree(dPixelColors));
}

