#include "hip/hip_runtime.h"
/**
 * File Name : device.cu
 * Author : Yang Fan
 * Date : 2019/4/2
 * define device mixed
 */

#pragma once
#include <>
#include "../common/device.h"
#include "../common/define.h"
#include "../common/texture.h"
#include <vector>

__device__ void intersect(const Ray& ray, const Triangle& triangle, IntersectResult& intersectResult) {
	const auto origin = ray.origin;
	const auto direction = ray.direction;

	const auto u = triangle.top.pos3D - triangle.btm.pos3D;
	const auto v = triangle.mid.pos3D - triangle.btm.pos3D;
	const auto norm = Math::Vector3::cross(u, v).normalize();

	const auto b = Math::Vector3::dot(norm, direction);

	if (fabs(b) < 0) return;

	const auto w0 = origin - triangle.btm.pos3D;

	const auto a = - Math::Vector3::dot(norm, w0);

	const auto r = a / b;
	if (r <= 0.0f) return;

	intersectResult.intersectPoint = origin + direction * r;
	intersectResult.distance = r;

	const auto uu = Math::Vector3::dot(u, u);
	const auto uv = Math::Vector3::dot(u, v);
	const auto vv = Math::Vector3::dot(v, v);

	const auto w = intersectResult.intersectPoint - triangle.btm.pos3D;

	const auto wu = Math::Vector3::dot(w, u);
	const auto wv = Math::Vector3::dot(w, v);

	const auto d = uv * uv - uu * vv;

	const auto s = (uv * wv - vv * wu) / d;
	if (s < 0.0f || s > 1.0f) return;

	const auto t = (uv * wu - uu * wv) / d;
	if (t < 0.0f || (s + t) > 1.0f) return;


	intersectResult.isSucceed = true;
}

/*
 * ������ɫ
 */
__device__ void SetPixel(int x, int y, const Color& color, Uint8* pixelColors, int screenWidth, int screenHeight) {
	auto r = color.r;
	auto g = color.g;
	auto b = color.b;
	auto a = color.a;
	CLAMP01(a);
	CLAMP01(b);
	CLAMP01(g);
	CLAMP01(r);

	auto i = (y * screenWidth + x) * 4;
	const auto size = (screenWidth - 1) * (screenHeight - 1) * 4;

	CLAMP(i , 4 , size);

	pixelColors[i - 1] = static_cast<Uint8>(a * 255);
	pixelColors[i - 2] = static_cast<Uint8>(b * 255);
	pixelColors[i - 3] = static_cast<Uint8>(g * 255);
	pixelColors[i - 4] = static_cast<Uint8>(r * 255);
}

/*
 * ��Ȳ���
 */
__device__ void TestDepth(int x, int y, float depth, float* depths, bool& isSuccess, int screenWidth,
                          int screenHeight) {
	if (x >= screenWidth || x <= 0 || y <= 0 || y >= screenHeight) return;

	const auto idx = y * screenWidth + x;
	const auto cdp = depths[idx];

	if (cdp == 0 || depth <= cdp) {
		depths[idx] = depth;
		isSuccess = true;
	}
}

/*
 * �������
 */
__device__ void SampleLight(Pixel& pixel, Triangle* triangles, Color& color, int numTriangles) {

	const auto lightDirection = Math::Vector3(0, 1, -1).normalize();
	const Math::Vector3 normal = pixel.normal.normalize();

	// ������Ӱ
	Ray ray;
	ray.isActive = true;
	ray.origin = pixel.pos3D;
	ray.direction = lightDirection;

	bool isShadow = false;
	
	for (auto i = 0; i < numTriangles; ++i) {
		IntersectResult iTmp{false};
		intersect(ray, triangles[i], iTmp);
		if (iTmp.isSucceed && iTmp.distance > 0.01f) {
			isShadow = true;
			break;
		}
	}

	// printf("%.2f %.2f %.2f\n" , ray.origin._x , ray.origin._y , ray.origin._z);

	// �������	
	float ambient = 0.2;
	
	float nd = Math::Vector3::dot(lightDirection, normal);
	CLAMP01(nd);

	color = color * (ambient + nd * !isShadow) + pixel.color * 0.01f;
}

/*
 * ���㷴��
 */
__device__ void SampleReflect() {

}


/*
 * ��Ⱦ���߻�Ͻ׶�
 */
__global__ void KernelMixed(Pixel* pixels, Color* colors, Triangle* triangles, Uint8* pixelColors, float* depths,
                            int screenWidth, int screenHeight, int numTriangles, int numElements) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numElements) {

		const int x = pixels[idx].pos._x;
		const int y = pixels[idx].pos._y;

		auto isFirst = false;
		TestDepth(x, y, pixels[idx].pos._z, depths, isFirst, screenWidth, screenHeight);
		if (isFirst) {

			SampleLight(pixels[idx], triangles, colors[idx], numTriangles);

			SetPixel(x, y, colors[idx], pixelColors, screenWidth, screenHeight);
		}
	}
}


extern "C" void CallMixed(std::vector<Pixel>& pixels, std::vector<Color>& colors,
                          const std::vector<Triangle>& triangles, Uint8* pixelColors, float* depths, int screenWidth,
                          int screenHeight) {
	if (pixels.empty()) return;

	const int numPixels = pixels.size();
	const int numTriangles = triangles.size();
	const int screenPixelSize = screenWidth * screenHeight;

	Pixel* dPixels;
	CUDA_CALL(hipMalloc(&dPixels , sizeof(Pixel) * numPixels));
	CUDA_CALL(hipMemset(dPixels , 0 , sizeof(Pixel) * numPixels));
	Color* dColors;
	CUDA_CALL(hipMalloc(&dColors , sizeof(Color) * numPixels));
	CUDA_CALL(hipMemset(dColors , 0 , sizeof(Color) * numPixels));
	Uint8* dPixelColors;
	CUDA_CALL(hipMalloc(&dPixelColors , sizeof(Uint8) * screenPixelSize * 4));
	CUDA_CALL(hipMemset(dPixelColors , 0 , sizeof(Uint8) * screenPixelSize * 4));
	float* dDepths;
	CUDA_CALL(hipMalloc(&dDepths , sizeof(float) * screenPixelSize));
	CUDA_CALL(hipMemset(dDepths , 0 , sizeof(float) * screenPixelSize));

	Triangle* dTriangles;
	CUDA_CALL(hipMalloc(&dTriangles , sizeof(Triangle) * numTriangles));
	CUDA_CALL(hipMemset(dTriangles , 0 , sizeof(Triangle) * numTriangles));


	CUDA_CALL(hipMemcpy(dPixelColors , pixelColors , sizeof(Uint8) * screenPixelSize * 4 , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dDepths , depths , sizeof(float) * screenPixelSize , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dPixels , &pixels[0] , sizeof(Pixel) * numPixels , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dColors , &colors[0] , sizeof(Color) * numPixels , hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dTriangles , &triangles[0] , sizeof(Triangle) * numTriangles , hipMemcpyHostToDevice));

	// ��ˮ��ִ��

	KernelMixed<<<(numPixels + 63) / 64 , 64>>>(dPixels, dColors, dTriangles, dPixelColors, dDepths, screenWidth,
	                                            screenHeight, numTriangles, numPixels);

	hipDeviceSynchronize();

	CUDA_CALL(hipMemcpy(depths , dDepths , sizeof(float) * screenPixelSize , hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(pixelColors , dPixelColors , sizeof(Uint8) * screenPixelSize * 4, hipMemcpyDeviceToHost));

	CUDA_CALL(hipFree(dDepths));
	CUDA_CALL(hipFree(dPixels));
	CUDA_CALL(hipFree(dColors));
	CUDA_CALL(hipFree(dPixelColors));
}

