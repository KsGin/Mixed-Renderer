#include "hip/hip_runtime.h"
/**
 * File Name : raster.h
 * Author : Yang Fan
 * Date : 2018/11/28
 * declare raster
 */

#pragma once

#include "../includes/math/vector.hpp"
#include "../common/define.h"
#include "../common/color.h"
#include <cmath>
#include <vector>

/*
 * INTERPOLATE FLOAT VALUE DEFINED
 */
#define INTERPOLATE(a , b , g , r) {								\
	CLAMP01(g);													\
	int d = a > b;													\
	r = d * (a - (a - b) * g) + (1-d) * (a + (b - a) * g);		\
}

/*
 * INTERPOLATE VECTOR2 VALUE DEFINED
 */
#define INTERPOLATEV2(v1 , v2 , gad , result) {								\
	INTERPOLATE(v1._x, v2._x, gad , result._x);								\
	INTERPOLATE(v1._y, v2._y, gad , result._y);								\
}

/*
 * INTERPOLATE VECTOR3 VALUE DEFINED
 */
#define INTERPOLATEV3(v1 , v2 , gad , result) {								\
	INTERPOLATE(v1._x, v2._x, gad , result._x);								\
	INTERPOLATE(v1._y, v2._y, gad , result._y);								\
	INTERPOLATE(v1._z, v2._z, gad , result._z);								\
}

/*
 * INTERPOLATE COLOR VALUE DEFINED
 */
#define INTERPOLATEC(v1 , v2 , gad , result) {								\
	INTERPOLATE(v1.r, v2.r, gad , result.r);								\
	INTERPOLATE(v1.g, v2.g, gad , result.g);								\
	INTERPOLATE(v1.b, v2.b, gad , result.b);								\
	INTERPOLATE(v1.a, v2.a, gad , result.a);								\
}

#define INTERPOLATEP(p1 , p2 , gad , result) {								\
	INTERPOLATEV3(p1.pos , p2.pos , gad , result.pos);						\
	INTERPOLATEV3(p1.normal , p2.normal , gad , result.normal);				\
	INTERPOLATEV2(p1.uv , p2.uv , gad , result.uv);							\
	INTERPOLATEC(p1.color , p2.color , gad , result.color);					\
}

class Raster {
private:
	/*
	 * Bresenham Line Algorithm
	 */
	static void RasterizeLine(const Pixel& p1, const Pixel& p2, std::vector<Pixel>& pixels) {

		auto start = p1, end = p2;

		if (p1.pos._x > p2.pos._x) {
			start = p2;
			end = p1;
		}

		float gad = 0.0f;
		float disx = abs(end.pos._x - start.pos._x);
		float disy = abs(end.pos._y - start.pos._y);
		float dis = disx > disy ? disx : disy;

		Pixel p;
		for (auto i = 0; i < dis; i++) {
			gad = i / dis;
			INTERPOLATEP(p1 , p2 , gad , p);
			pixels.emplace_back(p);
		}
	}

	static void RasterizeTriangle(const Pixel& top, const Pixel& mid, const Pixel& btm, std::vector<Pixel>& pixels) {
		for (auto y = top.pos._y; y >= btm.pos._y; --y) {
			Pixel sp, ep;
			float sgad = 0.0f, egad = 0.0f;
			if (y >= mid.pos._y) {
				sgad = (y - top.pos._y) / (mid.pos._y - top.pos._y);
				egad = (y - top.pos._y) / (btm.pos._y - top.pos._y);
				INTERPOLATEP(top , mid , sgad , sp);
				INTERPOLATEP(top , btm , egad , ep);
			}
			else {
				sgad = (y - mid.pos._y) / (btm.pos._y - mid.pos._y);
				egad = (y - top.pos._y) / (btm.pos._y - top.pos._y);
				INTERPOLATEP(mid , btm , sgad , sp);
				INTERPOLATEP(top , btm , egad , ep);
			}


			if (sp.pos._x > ep.pos._x) {
				auto tp = sp;
				sp = ep;
				ep = tp;
			}

			float sx = sp.pos._x, ex = ep.pos._x; // x����	

			Pixel p;
			for (auto x = sx; x <= ex; ++x) {
				float gad = 0;
				if (ex - sx < 1.0f) { gad = 0; }
				else gad = (x - sx) / (ex - sx);
				INTERPOLATEP(sp , ep , gad , p);
				pixels.emplace_back(p);
			}
		}
	}

public:

	/*
	 * ������ʱ���ݽṹ
	 */
	struct Triangle {
		/*
		 * ��������
		 */
		Pixel top, mid, btm;
		/*
		 * ���ظ���
		 */
		int numPixels;
	};

	static void rasterize(Triangle& triangle, std::vector<Pixel>& pixels, const TYPE type) {
		if (type == SOLID) {
			RasterizeTriangle(triangle.top, triangle.mid, triangle.btm, pixels);
		}
		else {
			RasterizeLine(triangle.top, triangle.mid, pixels);
			RasterizeLine(triangle.top, triangle.btm, pixels);
			RasterizeLine(triangle.mid, triangle.btm, pixels);
		}
	}
};
